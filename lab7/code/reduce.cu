#include "hip/hip_runtime.h"
#include "driver.h"
#include "reduce.h"
#include "cuda_utils.h"

dtype
reduceCpu (dtype* h_A, unsigned int N)
{
  int i;
  dtype ans;

  ans = (dtype) 0.0;
  for(i = 0; i < N; i++) {
    ans += h_A[i];
  }

  return ans;
}

__global__ void 
reduceNaiveKernel (dtype* In, dtype *Out, unsigned int N)
{
	__shared__ dtype buffer[BS];
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride;
	

	/* load data to buffer */
	if(tid < N) {
		buffer[threadIdx.x] = In[tid];
	} else {
		buffer[threadIdx.x] = (dtype) 0.0;
	}
	__syncthreads ();

	/* reduce in shared memory */
	for(stride = 1; stride < blockDim.x; stride *= 2) {
		if(threadIdx.x % (stride * 2) == 0) {
			buffer[threadIdx.x] += buffer[threadIdx.x + stride];
		}
		__syncthreads ();
	}

	/* store back the reduced result */
	if(threadIdx.x == 0) {
		Out[blockIdx.x] = buffer[0];
	}
}

dtype
reduceNaive (dtype* d_In, dtype* d_Out, dtype* h_Out, unsigned int N)
{
	unsigned int i, nThreads, tbSize, nBlocks;
	dtype ans;
	

	nThreads = N;
	tbSize = BS;
	nBlocks = (nThreads + tbSize - 1) / tbSize;

	dim3 grid (nBlocks);
	dim3 block (tbSize);

	for(i = 0; i < NUM_ITER; i++) {
		reduceNaiveKernel <<<grid, block>>> (d_In, d_Out, N);
		hipDeviceSynchronize ();
	}

	CUDA_CHECK_ERROR (hipMemcpy (h_Out, d_Out, nBlocks * sizeof (dtype),
																hipMemcpyDeviceToHost));

	ans = reduceCpu (h_Out, nBlocks);

	return ans;

}

__global__ void 
reduceNonDivergeKernel (dtype* In, dtype *Out, unsigned int N)
{
	__shared__ dtype buffer[BS];
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride;
	

	/* load data to buffer */
	if(tid < N) {
		buffer[threadIdx.x] = In[tid];
	} else {
		buffer[threadIdx.x] = (dtype) 0.0;
	}
	__syncthreads ();

	int num_threads = BS/2;
        /* reduce in shared memory */
        for(stride = 1; stride < blockDim.x; stride <<= 1) {
                if(threadIdx.x < num_threads)
                {
		        int pow = stride * 2 * threadIdx.x;
                        buffer[pow] += buffer[pow + stride];
                }
                __syncthreads();
	    num_threads >>= 1;
        }
	/* store back the reduced result */
	if(threadIdx.x == 0) {
		Out[blockIdx.x] = buffer[0];
	}
}



dtype
reduceNonDiverge (dtype* d_In, dtype* d_Out, dtype* h_Out, unsigned int N)
{
	unsigned int i, nThreads, tbSize, nBlocks;
	dtype ans;


	nThreads = N;
	tbSize = BS;
	nBlocks = (nThreads + tbSize - 1) / tbSize;

	dim3 grid (nBlocks);
	dim3 block (tbSize);

	for(i = 0; i < NUM_ITER; i++) {
		reduceNonDivergeKernel <<<grid, block>>> (d_In, d_Out, N);
		hipDeviceSynchronize ();
	}

	CUDA_CHECK_ERROR (hipMemcpy (h_Out, d_Out, nBlocks * sizeof (dtype),
																hipMemcpyDeviceToHost));

	ans = reduceCpu (h_Out, nBlocks);

	return ans;

}

__global__ void 
reduceSeqAddKernel (dtype* In, dtype *Out, unsigned int N)
{
	__shared__ dtype buffer[BS];
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	/* load data to buffer */
	if(tid < N) {
		buffer[threadIdx.x] = In[tid];
	} else {
		buffer[threadIdx.x] = (dtype) 0.0;
	}
	__syncthreads ();
	int num_threads = BS/2;
        /* reduce in shared memory */
        for(; num_threads > 0; num_threads >>= 1) {
            if( threadIdx.x < num_threads)
	    {
	    	buffer[threadIdx.x] += buffer[threadIdx.x + num_threads];
	        __syncthreads();
	    }
        }
	/* store back the reduced result */
	if(threadIdx.x == 0) {
		Out[blockIdx.x] = buffer[0];
	}
}



dtype
reduceSeqAdd (dtype* d_In, dtype* d_Out, dtype* h_Out, unsigned int N)
{
	unsigned int i, nThreads, tbSize, nBlocks;
	dtype ans;


	nThreads = N;
	tbSize = BS;
	nBlocks = (nThreads + tbSize - 1) / tbSize;

	dim3 grid (nBlocks);
	dim3 block (tbSize);

	for(i = 0; i < NUM_ITER; i++) {
		reduceSeqAddKernel <<<grid, block>>> (d_In, d_Out, N);
		hipDeviceSynchronize ();
	}

	CUDA_CHECK_ERROR (hipMemcpy (h_Out, d_Out, nBlocks * sizeof (dtype),
																hipMemcpyDeviceToHost));

	ans = reduceCpu (h_Out, nBlocks);

	return ans;

}

__global__ void 
reduceFirstAddKernel (dtype* In, dtype *Out, unsigned int N)
{
	/* As it can be seen from `reduceSeqAdd`, the total number of threads
		 have been halved */
	/* Thus, you need to load 2 elements from the global memory, add them, and
		 then store the sum in the shared memory before reduction over the shared
		 memory occurs */
	__shared__ dtype buffer[BS];
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	/* load data to buffer */
	if(tid < (N+1)/2) {
		buffer[threadIdx.x] = In[tid] + In[tid + N/2];
	} else {
		buffer[threadIdx.x] = (dtype) 0.0;
	}
	__syncthreads ();
	int num_threads = BS/2;
        /* reduce in shared memory */
        for(; num_threads > 0; num_threads >>= 1) {
            if( threadIdx.x < num_threads)
	    {
	    	buffer[threadIdx.x] += buffer[threadIdx.x + num_threads];
	        __syncthreads();
	    }
        }
	/* store back the reduced result */
	if(threadIdx.x == 0) {
		Out[blockIdx.x] = buffer[0];
	}
}



dtype
reduceFirstAdd (dtype* d_In, dtype* d_Out, dtype* h_Out, unsigned int N)
{
	unsigned int i, nThreads, tbSize, nBlocks;
	dtype ans;


	nThreads = (N + 1) / 2;
	tbSize = BS;
	nBlocks = (nThreads + tbSize - 1) / tbSize;

	dim3 grid (nBlocks);
	dim3 block (tbSize);

	for(i = 0; i < NUM_ITER; i++) {
		reduceFirstAddKernel <<<grid, block>>> (d_In, d_Out, N);
		hipDeviceSynchronize ();
	}

	CUDA_CHECK_ERROR (hipMemcpy (h_Out, d_Out, nBlocks * sizeof (dtype),
																hipMemcpyDeviceToHost));

	ans = reduceCpu (h_Out, nBlocks);

	return ans;

}

__global__ void 
reduceUnrollLastKernel (dtype* In, dtype *Out, unsigned int N)
{
	/* Fill in your code here */
	/* unroll the loop when there are fewer than 32 threads working */
	__shared__ dtype buffer[BS];
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	/* load data to buffer */
	if(tid < (N+1)/2) {
		buffer[threadIdx.x] = In[tid] + In[tid + N/2];
	} else {
		buffer[threadIdx.x] = (dtype) 0.0;
	}
	__syncthreads ();
	int num_threads = BS/2;
        /* reduce in shared memory */
        for(; num_threads > 32; num_threads >>= 1) {
            if( threadIdx.x < num_threads)
	    {
	    	buffer[threadIdx.x] += buffer[threadIdx.x + num_threads];
	        __syncthreads();
	    }
        }

	volatile dtype *sm = buffer;
	if(threadIdx.x < 32)
	{
		sm[threadIdx.x] += sm[threadIdx.x + num_threads];
		sm[threadIdx.x] += sm[threadIdx.x + num_threads/2];
		sm[threadIdx.x] += sm[threadIdx.x + num_threads/4];
		sm[threadIdx.x] += sm[threadIdx.x + num_threads/8];
		sm[threadIdx.x] += sm[threadIdx.x + num_threads/16];
		sm[threadIdx.x] += sm[threadIdx.x + num_threads/32];
	}	

	/* store back the reduced result */
	if(threadIdx.x == 0) {
		Out[blockIdx.x] = sm[0];
	}
}



dtype
reduceUnrollLast (dtype* d_In, dtype* d_Out, dtype* h_Out, unsigned int N)
{
	unsigned int i, nThreads, tbSize, nBlocks;
	dtype ans;


	nThreads = (N + 1) / 2;
	tbSize = BS;
	nBlocks = (nThreads + tbSize - 1) / tbSize;

	dim3 grid (nBlocks);
	dim3 block (tbSize);

	for(i = 0; i < NUM_ITER; i++) {
		reduceUnrollLastKernel <<<grid, block>>> (d_In, d_Out, N);
		hipDeviceSynchronize ();
	}

	CUDA_CHECK_ERROR (hipMemcpy (h_Out, d_Out, nBlocks * sizeof (dtype),
																hipMemcpyDeviceToHost));

	ans = reduceCpu (h_Out, nBlocks);

	return ans;

}

__global__ void 
reduceUnrollAllKernel (dtype* In, dtype *Out, unsigned int N)
{
	/* Fill in your code here */
	__shared__ dtype buffer[BS];
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	/* load data to buffer */
	if(tid < (N+1)/2) {
		buffer[threadIdx.x] = In[tid] + In[tid + N/2];
	} else {
		buffer[threadIdx.x] = (dtype) 0.0;
	}
	__syncthreads ();

	if(BS >= 1024) {
		if(threadIdx.x < 512) {buffer[threadIdx.x] += buffer[threadIdx.x + 512]; __syncthreads();}
	}

	if(BS >= 512) {
		if(threadIdx.x < 256) {buffer[threadIdx.x] += buffer[threadIdx.x + 256]; __syncthreads();}
	}

	if(BS >= 256) {
		if(threadIdx.x < 128) {buffer[threadIdx.x] += buffer[threadIdx.x + 128]; __syncthreads ();}
	}

	if(BS >= 128) {
		if(threadIdx.x < 64) {buffer[threadIdx.x] += buffer[threadIdx.x + 64]; __syncthreads ();}
	}


	volatile dtype *sm = buffer;
	if(threadIdx.x < 32)
	{
		sm[threadIdx.x] += sm[threadIdx.x + 32];
		sm[threadIdx.x] += sm[threadIdx.x + 16];
		sm[threadIdx.x] += sm[threadIdx.x + 8];
		sm[threadIdx.x] += sm[threadIdx.x + 4];
		sm[threadIdx.x] += sm[threadIdx.x + 2];
		sm[threadIdx.x] += sm[threadIdx.x + 1];
	}	

	/* store back the reduced result */
	if(threadIdx.x == 0) {
		Out[blockIdx.x] = sm[0];
	}
}



dtype
reduceUnrollAll (dtype* d_In, dtype* d_Out, dtype* h_Out, unsigned int N)
{
	unsigned int i, nThreads, tbSize, nBlocks;
	dtype ans;


	nThreads = (N + 1) / 2;
	tbSize = BS;
	nBlocks = (nThreads + tbSize - 1) / tbSize;

	dim3 grid (nBlocks);
	dim3 block (tbSize);

	for(i = 0; i < NUM_ITER; i++) {
		reduceUnrollAllKernel <<<grid, block>>> (d_In, d_Out, N);
		hipDeviceSynchronize ();
	}

	CUDA_CHECK_ERROR (hipMemcpy (h_Out, d_Out, nBlocks * sizeof (dtype),
																hipMemcpyDeviceToHost));

	ans = reduceCpu (h_Out, nBlocks);

	return ans;

}



__global__ void 
reduceMultAddKernel (dtype* In, dtype *Out, unsigned int N)
{
	/* Fill in your code here */
	/* Instead of just adding 2 elements in the beginning, try adding more 
		 before reducing the partial sums over the shared memory */
	__shared__ dtype buffer[BS];
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	/* load data to buffer */
	if(tid < (N+1)/32) {
		buffer[threadIdx.x] = In[tid] + In[tid + 1*(N/32)]+ In[tid + 2*(N/32)] + In[tid + 3*(N/32)] 
				     + In[tid + 4*(N/32)] + In[tid + 5*(N/32)] + In[tid + 6*(N/32)] + In[tid + 7*(N/32)]
				     + In[tid + 8*(N/32)] + In[tid + 9*(N/32)] + In[tid + 10*(N/32)] + In[tid + 11*(N/32)] 
				     + In[tid + 12*(N/32)] + In[tid + 13*(N/32)] + In[tid + 14*(N/32)] + In[tid + 15*(N/32)] 
				     + In[tid + 16*(N/32)] + In[tid + 17*(N/32)] + In[tid + 18*(N/32)] + In[tid + 19*(N/32)] 
				     + In[tid + 20*(N/32)] + In[tid + 21*(N/32)] + In[tid + 22*(N/32)] + In[tid + 23*(N/32)] 
				     + In[tid + 24*(N/32)] + In[tid + 25*(N/32)] + In[tid + 26*(N/32)] + In[tid + 27*(N/32)] 
				     + In[tid + 28*(N/32)] + In[tid + 29*(N/32)] + In[tid + 30*(N/32)] + In[tid + 31*(N/32)];

	//	for(int i=0; i<32; ++i){
	//		buffer[threadIdx.x] += In[tid + i*((N+1)/32)];
	//	}
	} else {
		buffer[threadIdx.x] = (dtype) 0.0;
	}
	__syncthreads ();

	if(BS >= 1024) {
		if(threadIdx.x < 512) {buffer[threadIdx.x] += buffer[threadIdx.x + 512]; __syncthreads();}
	}

	if(BS >= 512) {
		if(threadIdx.x < 256) {buffer[threadIdx.x] += buffer[threadIdx.x + 256]; __syncthreads();}
	}

	if(BS >= 256) {
		if(threadIdx.x < 128) {buffer[threadIdx.x] += buffer[threadIdx.x + 128]; __syncthreads ();}
	}

	if(BS >= 128) {
		if(threadIdx.x < 64) {buffer[threadIdx.x] += buffer[threadIdx.x + 64]; __syncthreads ();}
	}


	volatile dtype *sm = buffer;
	if(threadIdx.x < 32)
	{
		sm[threadIdx.x] += sm[threadIdx.x + 32];
		sm[threadIdx.x] += sm[threadIdx.x + 16];
		sm[threadIdx.x] += sm[threadIdx.x + 8];
		sm[threadIdx.x] += sm[threadIdx.x + 4];
		sm[threadIdx.x] += sm[threadIdx.x + 2];
		sm[threadIdx.x] += sm[threadIdx.x + 1];
	}	

	/* store back the reduced result */
	if(threadIdx.x == 0) {
		Out[blockIdx.x] = sm[0];
	}

}



dtype
reduceMultAdd (dtype* d_In, dtype* d_Out, dtype* h_Out, unsigned int N)
{
	unsigned int i, nThreads, tbSize, nBlocks;
	dtype ans;


	nThreads = (N + 1) / 32;
	tbSize = BS;
	nBlocks = (nThreads + tbSize - 1) / tbSize;

	dim3 grid (nBlocks);
	dim3 block (tbSize);

	for(i = 0; i < NUM_ITER; i++) {
		reduceMultAddKernel <<<grid, block>>> (d_In, d_Out, N);
		hipDeviceSynchronize ();
	}

	CUDA_CHECK_ERROR (hipMemcpy (h_Out, d_Out, nBlocks * sizeof (dtype),
																hipMemcpyDeviceToHost));

	ans = reduceCpu (h_Out, nBlocks);

	return ans;

}





void
initCudaArray (dtype **d_A, dtype *h_A, unsigned int N)
{
	CUDA_CHECK_ERROR (hipMalloc ((void**) d_A, N * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMemcpy (*d_A, h_A, N * sizeof (dtype),
																hipMemcpyHostToDevice));
}

void
cudaReduction (dtype *A, unsigned int N, unsigned int OPT, dtype *ret)
{
	dtype *h_Out, *d_Out;
	unsigned int nBlocks;

	hipEvent_t start, stop;
	float elapsedTime;

	dtype ans;

	nBlocks = (N + BS - 1) / BS;
	h_Out = (dtype*) malloc (nBlocks * sizeof (dtype));
	CUDA_CHECK_ERROR (hipMalloc ((void**) &d_Out, nBlocks * sizeof (dtype)));
	
	CUDA_CHECK_ERROR (hipEventCreate (&start));
	CUDA_CHECK_ERROR (hipEventCreate (&stop));

	fprintf (stderr, "Executing test case [%d]\n", OPT);
	fprintf (stderr, "[1]: Naive | [2]: Non-divergent | [3]: Sequential Add. | [4]: First add | [5]: Unroll last warp | [6]: Complete unroll | [7] Multiple Adds\n");

	
	CUDA_CHECK_ERROR (hipEventRecord (start, 0));
	/* execute kernel */
	switch (OPT) {
		case 1:
			ans = reduceNaive (A, d_Out, h_Out, N);	
			break;
		case 2:
			ans = reduceNonDiverge (A, d_Out, h_Out, N);	
			break;
		case 3:
			ans = reduceSeqAdd (A, d_Out, h_Out, N);	
			break;
		case 4:
			ans = reduceFirstAdd (A, d_Out, h_Out, N);	
			break;
		case 5:
			ans = reduceUnrollLast (A, d_Out, h_Out, N);	
			break;
		case 6:
			ans = reduceUnrollAll (A, d_Out, h_Out, N);	
			break;
		case 7:
			ans = reduceMultAdd (A, d_Out, h_Out, N);	
			break;
		default:
			ans = reduceNaive (A, d_Out, h_Out, N);	
	} 
	CUDA_CHECK_ERROR (hipEventRecord (stop, 0));
	CUDA_CHECK_ERROR (hipEventSynchronize (stop));
	CUDA_CHECK_ERROR (hipEventElapsedTime (&elapsedTime, start, stop));
	elapsedTime = elapsedTime / NUM_ITER;


	fprintf (stderr, "Execution time: %f ms\n", elapsedTime);
	fprintf (stderr, "Equivalent performance: %f GB/s\n", 
						(N * sizeof (dtype) / elapsedTime) * 1e-6);

	CUDA_CHECK_ERROR (hipEventDestroy (start));
	CUDA_CHECK_ERROR (hipEventDestroy (stop));

	free (h_Out);
	CUDA_CHECK_ERROR (hipFree (d_Out));

	*ret = ans;	
}


