#include "hip/hip_runtime.h"
#include "driver.h"
#include "mm.h"
#include "cuda_utils.h"

void
initCudaArray (dtype **d_A, dtype *h_A, unsigned int N)
{
	CUDA_CHECK_ERROR (hipMalloc ((void**) d_A, N * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMemcpy (*d_A, h_A, N * sizeof (dtype),
																hipMemcpyHostToDevice));
}


__global__
void
mmSharedKernel (dtype* A, dtype* B, dtype* C, unsigned int N)
{

	

	/* block indices */
	int bidx = blockIdx.x;
	int bidy = blockIdx.y;

	/* thread indices */
	int tidx = threadIdx.x;
	int tidy = threadIdx.y;

	/* row  index of first sub-block of matrix A processed by this thread block */
	int aStart = N * (BLOCK_SIZE * bidy);
	/* row  index of last sub-block of matrix A processed by this thread block */
	int aEnd   = aStart + N - 1;
	/* increment size for sub-block of matrix A */
	int aInc = BLOCK_SIZE;

	/* col index of first sub-blcok of matrx B processed by this thread block */
	int bStart = BLOCK_SIZE * bidx;
	/* last sub block is not needed since it'll have 1-on-1 match to A */
	/* increment size for sub-block of matrix B */
	int bInc = BLOCK_SIZE * N;

	/* temporary variable for accummulating the partial results */
	float cSub = 0;

	/* Loop over the sub-matrices of A and B */
	for (int a = aStart, b = bStart; a <= aEnd; a += aInc, b += bInc) {
		/* declaration of shared memory for storing sub-block of A */
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

		/* declaration of shared memory for storing sub-block of B */
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		/* load the matrices from memory to shared memory */
		As[tidy][tidx] = A[a + N * tidy + tidx];
		Bs[tidy][tidx] = B[b + N * tidy + tidx];
		__syncthreads();

		/* multiply the two matrices together */
		/* one thread per element of C */
#pragma unroll
		for (int k = 0; k < BLOCK_SIZE; ++k)
			cSub += As[tidy][k] * Bs[k][tidx];

		/* synchornize before loading next sub-blocks */
		__syncthreads();
	}

	/* write back the results */
	int c = N * BLOCK_SIZE * bidy + BLOCK_SIZE * bidx;
	C[c + N * tidy + tidx] = cSub;

}
void
mmShared (dtype* A, dtype* B, dtype* C, unsigned int N)
{
	unsigned int nBlocks;


	nBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

	dim3 grid (nBlocks, nBlocks);	
	dim3 block (BLOCK_SIZE, BLOCK_SIZE);	

	mmSharedKernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmSharedKernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmSharedKernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmSharedKernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmSharedKernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
}



__global__
void
mmNaiveKernel (dtype* A, dtype* B, dtype* C, unsigned int N)
{
	int i;
	dtype sum;
	int gidx = threadIdx.x + blockIdx.x * blockDim.x; /* column (j) */
	int gidy = threadIdx.y + blockIdx.y * blockDim.y; /* row (i) */
	int gid = gidx + gidy * N;

	sum = 0.0;
	for(i = 0; i < N; i++) {
		sum += A[gidy * N + i] * B[i * N + gidx];
	}
	C[gid] = sum;
}
void
mmNaive (dtype* A, dtype* B, dtype* C, unsigned int N)
{
	unsigned int nBlocks;


	nBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

	dim3 grid (nBlocks, nBlocks);	
	dim3 block (BLOCK_SIZE, BLOCK_SIZE);	


	mmNaiveKernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmNaiveKernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmNaiveKernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmNaiveKernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();


	mmNaiveKernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
}


__global__
void
mmShared2Kernel (dtype* A, dtype* B, dtype* C, unsigned int N)
{
        /* block indices */
        int bidx = blockIdx.x;
        int bidy = blockIdx.y;

        /* thread indices */
        int tidx = threadIdx.x;
        int tidy = threadIdx.y;

        /* row  index of first sub-block of matrix A processed by this thread block */
        int aStart = N * (BLOCK_SIZE * bidy);
        /* row  index of last sub-block of matrix A processed by this thread block */
        int aEnd   = aStart + N - 1;
        /* increment size for sub-block of matrix A */
        int aInc = BLOCK_SIZE;

        /* col index of first sub-blcok of matrx B processed by this thread block */
        int bStart = BLOCK_SIZE * bidx;
        /* last sub block is not needed since it'll have 1-on-1 match to A */
        /* increment size for sub-block of matrix B */
        int bInc = BLOCK_SIZE * N;


	/* temporary variable for accummulating the partial results */
        float cSub = 0;
        float cSub_1 = 0;

        int incr = BLOCK_SIZE >> 1;
        /* Loop over the sub-matrices of A and B */
        for (int a = aStart, b = bStart; a <= aEnd; a += aInc, b += bInc) {
                /* declaration of shared memory for storing sub-block of A */
                __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

                /* declaration of shared memory for storing sub-block of B */
                __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

                /* load the matrices from memory to shared memory */
                As[tidy][tidx] = A[a + N * tidy + tidx];
                As[tidy + incr][tidx] = A[a + N * (tidy + incr) + tidx];

                Bs[tidy][tidx] = B[b + N * tidy + tidx];
                Bs[tidy + incr][tidx]  = B[b + N * (tidy + incr) + tidx];

                __syncthreads();

                /* multiply the two matrices together */
                /* one thread per element of C */
#pragma unroll
                for (int k = 0; k < BLOCK_SIZE; ++k){
                        cSub += As[tidy][k] * Bs[k][tidx];
                        cSub_1 += As[tidy + incr][k] * Bs[k][tidx];
                }
                /* synchornize before loading next sub-blocks */
                __syncthreads();
        }

        /* write back the results */
        int c = N * BLOCK_SIZE * bidy + BLOCK_SIZE * bidx;
        C[c + N * tidy + tidx] = cSub;
        C[c + N * (tidy + incr) + tidx] = cSub_1;

}
void
mmShared2 (dtype* A, dtype* B, dtype* C, unsigned int N)
{
	unsigned int nBlocks;


	nBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

	dim3 grid (nBlocks, nBlocks);	
	dim3 block (BLOCK_SIZE, BLOCK_SIZE / 2);	

	mmShared2Kernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmShared2Kernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmShared2Kernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmShared2Kernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmShared2Kernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
}


__global__
void
mmShared4Kernel (dtype* A, dtype* B, dtype* C, unsigned int N)
{
	
	/* block indices */
        int bidx = blockIdx.x;
        int bidy = blockIdx.y;

        /* thread indices */
        int tidx = threadIdx.x;
        int tidy = threadIdx.y;

        /* row  index of first sub-block of matrix A processed by this thread block */
        int aStart = N * (BLOCK_SIZE * bidy);
        /* row  index of last sub-block of matrix A processed by this thread block */
        int aEnd   = aStart + N - 1;
        /* increment size for sub-block of matrix A */
        int aInc = BLOCK_SIZE;

        /* col index of first sub-blcok of matrx B processed by this thread block */
        int bStart = BLOCK_SIZE * bidx;
        /* last sub block is not needed since it'll have 1-on-1 match to A */
        /* increment size for sub-block of matrix B */
        int bInc = BLOCK_SIZE * N;


	/* temporary variable for accummulating the partial results */
        float cSub = 0;
        float cSub_1 = 0;
	float cSub_2 = 0;
	float cSub_3 = 0;

        int incr = BLOCK_SIZE >> 2;
        /* Loop over the sub-matrices of A and B */
        for (int a = aStart, b = bStart; a <= aEnd; a += aInc, b += bInc) {
                /* declaration of shared memory for storing sub-block of A */
                __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

                /* declaration of shared memory for storing sub-block of B */
                __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

                /* load the matrices from memory to shared memory */
                As[tidy][tidx] = A[a + N * tidy + tidx];
                As[tidy + incr][tidx] = A[a + N * (tidy + incr) + tidx];
                As[tidy + 2*incr][tidx] = A[a + N * (tidy + 2*incr) + tidx];
                As[tidy + 3*incr][tidx] = A[a + N * (tidy + 3*incr) + tidx];

                Bs[tidy][tidx] = B[b + N * tidy + tidx];
                Bs[tidy + incr][tidx]  = B[b + N * (tidy + incr) + tidx];
                Bs[tidy + 2*incr][tidx]  = B[b + N * (tidy + 2*incr) + tidx];
                Bs[tidy + 3*incr][tidx]  = B[b + N * (tidy + 3*incr) + tidx];

                __syncthreads();

                /* multiply the two matrices together */
                /* one thread per element of C */
#pragma unroll
                for (int k = 0; k < BLOCK_SIZE; ++k){
                        cSub += As[tidy][k] * Bs[k][tidx];
                        cSub_1 += As[tidy + incr][k] * Bs[k][tidx];
			cSub_2 += As[tidy + 2*incr][k] * Bs[k][tidx];
			cSub_3 += As[tidy + 3*incr][k] * Bs[k][tidx];
                }
                /* synchornize before loading next sub-blocks */
                __syncthreads();
        }

        /* write back the results */
        int c = N * BLOCK_SIZE * bidy + BLOCK_SIZE * bidx;
        C[c + N * tidy + tidx] = cSub;
        C[c + N * (tidy + incr) + tidx] = cSub_1;
        C[c + N * (tidy + 2*incr) + tidx] = cSub_2;
        C[c + N * (tidy + 3*incr) + tidx] = cSub_3;

}
void
mmShared4 (dtype* A, dtype* B, dtype* C, unsigned int N)
{
	unsigned int nBlocks;


	nBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

	dim3 grid (nBlocks, nBlocks);	
	dim3 block (BLOCK_SIZE, BLOCK_SIZE / 4);	

	mmShared4Kernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmShared4Kernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmShared4Kernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmShared4Kernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmShared4Kernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
}



__global__
void
mmShared8Kernel (dtype* A, dtype* B, dtype* C, unsigned int N)
{
        /* block indices */
        int bidx = blockIdx.x;
        int bidy = blockIdx.y;

        /* thread indices */
        int tidx = threadIdx.x;
        int tidy = threadIdx.y;

        /* row  index of first sub-block of matrix A processed by this thread block */
        int aStart = N * (BLOCK_SIZE * bidy);
        /* row  index of last sub-block of matrix A processed by this thread block */
        int aEnd   = aStart + N - 1;
        /* increment size for sub-block of matrix A */
        int aInc = BLOCK_SIZE;

        /* col index of first sub-blcok of matrx B processed by this thread block */
        int bStart = BLOCK_SIZE * bidx;
        /* last sub block is not needed since it'll have 1-on-1 match to A */
        /* increment size for sub-block of matrix B */
        int bInc = BLOCK_SIZE * N;


        /* temporary variable for accummulating the partial results */
        float cSub = 0;
	float cSub_1 = 0;
	float cSub_2 = 0;
	float cSub_3 = 0;
	float cSub_4 = 0;
	float cSub_5 = 0;
	float cSub_6 = 0;
	float cSub_7 = 0;
	

        int incr = BLOCK_SIZE >> 3;
        /* Loop over the sub-matrices of A and. B */
        for (int a = aStart, b = bStart; a <= aEnd; a += aInc, b += bInc) {
                /* declaration of shared memory for storing sub-block of A */
                __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
	

                /* declaration of shared memory for storing sub-block of B */
                __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

                /* load the matrices from memory to shared memory */
                As[tidy][tidx] = A[a + N * tidy + tidx];
                As[tidy + incr][tidx] = A[a + N * (tidy + incr) + tidx];
                As[tidy + 2*incr][tidx] = A[a + N * (tidy + 2*incr) + tidx];
                As[tidy + 3*incr][tidx] = A[a + N * (tidy + 3*incr) + tidx];
                As[tidy + 4*incr][tidx] = A[a + N * (tidy + 4*incr) + tidx];
                As[tidy + 5*incr][tidx] = A[a + N * (tidy + 5*incr) + tidx];
                As[tidy + 6*incr][tidx] = A[a + N * (tidy + 6*incr) + tidx];
                As[tidy + 7*incr][tidx] = A[a + N * (tidy + 7*incr) + tidx];


                Bs[tidy][tidx] = B[b + N * tidy + tidx];
                Bs[tidy + incr][tidx]  = B[b + N * (tidy + incr) + tidx];
                Bs[tidy + 2*incr][tidx]  = B[b + N * (tidy + 2*incr) + tidx];
                Bs[tidy + 3*incr][tidx]  = B[b + N * (tidy + 3*incr) + tidx];
                Bs[tidy + 4*incr][tidx]  = B[b + N * (tidy + 4*incr) + tidx];
                Bs[tidy + 5*incr][tidx]  = B[b + N * (tidy + 5*incr) + tidx];
                Bs[tidy + 6*incr][tidx]  = B[b + N * (tidy + 6*incr) + tidx];
                Bs[tidy + 7*incr][tidx]  = B[b + N * (tidy + 7*incr) + tidx];

                __syncthreads();

                /* multiply the two matrices together */
                /* one thread per element of C */
#pragma unroll
                for (int k = 0; k < BLOCK_SIZE; ++k){
                        cSub += As[tidy][k] * Bs[k][tidx];
                        cSub_1 += As[tidy + incr][k] * Bs[k][tidx];
                        cSub_2 += As[tidy + 2*incr][k] * Bs[k][tidx];
                        cSub_3 += As[tidy + 3*incr][k] * Bs[k][tidx];
                        cSub_4 += As[tidy + 4*incr][k] * Bs[k][tidx];
                        cSub_5 += As[tidy + 5*incr][k] * Bs[k][tidx];
                        cSub_6 += As[tidy + 6*incr][k] * Bs[k][tidx];
                        cSub_7 += As[tidy + 7*incr][k] * Bs[k][tidx];
                }
                /* synchornize before loading next sub-blocks */
                __syncthreads();
        }

        /* write back the results */
        int c = N * BLOCK_SIZE * bidy + BLOCK_SIZE * bidx;
        C[c + N * tidy + tidx] = cSub;
        C[c + N * (tidy + incr) + tidx] = cSub_1;
        C[c + N * (tidy + 2*incr) + tidx] = cSub_2;
        C[c + N * (tidy + 3*incr) + tidx] = cSub_3;
        C[c + N * (tidy + 4*incr) + tidx] = cSub_4;
        C[c + N * (tidy + 5*incr) + tidx] = cSub_5;
        C[c + N * (tidy + 6*incr) + tidx] = cSub_6;
        C[c + N * (tidy + 7*incr) + tidx] = cSub_7;

}
void
mmShared8 (dtype* A, dtype* B, dtype* C, unsigned int N)
{
	unsigned int nBlocks;


	nBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

	dim3 grid (nBlocks, nBlocks);	
	dim3 block (BLOCK_SIZE, BLOCK_SIZE / 8);	

	mmShared8Kernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmShared8Kernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmShared8Kernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmShared8Kernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmShared8Kernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
}

__global__
void
mmMyOwnKernel (dtype* A, dtype* B, dtype* C, unsigned int N)
{
	/* block indices */
        int bidx = blockIdx.x;
        int bidy = blockIdx.y;

        /* thread indices */
        int tidx = threadIdx.x;
        int tidy = threadIdx.y;

        /* row  index of first sub-block of matrix A processed by this thread block */
        int aStart = N * (BLOCK_SIZE * bidy);
        /* row  index of last sub-block of matrix A processed by this thread block */
        int aEnd   = aStart + N - 1;
        /* increment size for sub-block of matrix A */
        int aInc = BLOCK_SIZE;

        /* col index of first sub-blcok of matrx B processed by this thread block */
        int bStart = BLOCK_SIZE * bidx;
        /* last sub block is not needed since it'll have 1-on-1 match to A */
        /* increment size for sub-block of matrix B */
        int bInc = BLOCK_SIZE * N;


        /* temporary variable for accummulating the partial results */
        float cSub = 0;
        float cSub_1 = 0;
        float cSub_2 = 0;
        float cSub_3 = 0;
        float cSub_4 = 0;
        float cSub_5 = 0;
        float cSub_6 = 0;
        float cSub_7 = 0;
        float cSub_8 = 0;
        float cSub_9 = 0;
        float cSub_10 = 0;
        float cSub_11 = 0;
        float cSub_12 = 0;
        float cSub_13 = 0;
        float cSub_14 = 0;
	float cSub_15 = 0;

        int y_incr = BLOCK_SIZE >> 3;
	int x_incr = BLOCK_SIZE >> 1;
        /* Loop over the sub-matrices of A and B */
        for (int a = aStart, b = bStart; a <= aEnd; a += aInc, b += bInc) {
                /* declaration of shared memory for storing sub-block of A */
                __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];


                /* declaration of shared memory for storing sub-block of B */
                __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

                /* load the matrices from memory to shared memory */

		for(int i=0; i<8; i++){
			As[tidy + i*y_incr][tidx] = A[a + N * (tidy + i * y_incr) + tidx];
			As[tidy + i*y_incr][tidx + x_incr] = A[a + N * (tidy + i * y_incr) + tidx + x_incr];
		}	
/*
                As[tidy][tidx] = A[a + N * tidy + tidx];
                As[tidy][tidx + x_incr] = A[a + N * tidy + tidx + x_incr];
                
		As[tidy + y_incr][tidx] = A[a + N * (tidy + y_incr) + tidx];
                As[tidy + y_incr][tidx + x_incr] = A[a + N * (tidy + y_incr) + tidx + x_incr];
                
		As[tidy + 2*y_incr][tidx] = A[a + N * (tidy + 2*y_incr) + tidx];
                As[tidy + 2*y_incr][tidx + x_incr] = A[a + N * (tidy + 2*y_incr) + tidx + x_incr];
                
		As[tidy + 3*y_incr][tidx] = A[a + N * (tidy + 3*y_incr) + tidx];
                As[tidy + 3*y_incr][tidx + x_incr] = A[a + N * (tidy + 3*y_incr) + tidx + x_incr];
                
		As[tidy + 4*y_incr][tidx] = A[a + N * (tidy + 4*y_incr) + tidx];
                As[tidy + 4*y_incr][tidx + x_incr] = A[a + N * (tidy + 4*y_incr) + tidx + x_incr];
                
		As[tidy + 5*y_incr][tidx] = A[a + N * (tidy + 5*y_incr) + tidx];
                As[tidy + 5*y_incr][tidx + x_incr] = A[a + N * (tidy + 5*y_incr) + tidx + x_incr];
                
		As[tidy + 6*y_incr][tidx] = A[a + N * (tidy + 6*y_incr) + tidx];
                As[tidy + 6*y_incr][tidx + x_incr] = A[a + N * (tidy + 6*y_incr) + tidx + x_incr];
                
		As[tidy + 7*y_incr][tidx] = A[a + N * (tidy + 7*y_incr) + tidx];
                As[tidy + 7*y_incr][tidx + x_incr] = A[a + N * (tidy + 7*y_incr) + tidx + x_incr];
              
*/		
		for(int j=0; j<8; j++){
			Bs[tidy + j * y_incr][tidx] = B[b + N * (tidy + j * y_incr) + tidx];
			Bs[tidy + j * y_incr][tidx + x_incr] = B[b + N * (tidy + j * y_incr) + tidx + x_incr];
		}
/*

                Bs[tidy][tidx] = B[b + N * tidy + tidx];
                Bs[tidy][tidx + x_incr] = B[b + N * tidy + tidx + x_incr];
                
		Bs[tidy + y_incr][tidx]  = B[b + N * (tidy + y_incr) + tidx];
                Bs[tidy + y_incr][tidx + x_incr]  = B[b + N * (tidy + y_incr) + tidx + x_incr];
                
		Bs[tidy + 2*y_incr][tidx]  = B[b + N * (tidy + 2*y_incr) + tidx];
                Bs[tidy + 2*y_incr][tidx + x_incr]  = B[b + N * (tidy + 2*y_incr) + tidx + x_incr];
                
		Bs[tidy + 3*y_incr][tidx]  = B[b + N * (tidy + 3*y_incr) + tidx];
                Bs[tidy + 3*y_incr][tidx + x_incr]  = B[b + N * (tidy + 3*y_incr) + tidx + x_incr];
                
		Bs[tidy + 4*y_incr][tidx]  = B[b + N * (tidy + 4*y_incr) + tidx];
                Bs[tidy + 4*y_incr][tidx + x_incr]  = B[b + N * (tidy + 4*y_incr) + tidx + x_incr];
                
		Bs[tidy + 5*y_incr][tidx]  = B[b + N * (tidy + 5*y_incr) + tidx];
                Bs[tidy + 5*y_incr][tidx + x_incr]  = B[b + N * (tidy + 5*y_incr) + tidx + x_incr];
                
		Bs[tidy + 6*y_incr][tidx]  = B[b + N * (tidy + 6*y_incr) + tidx];
                Bs[tidy + 6*y_incr][tidx + x_incr]  = B[b + N * (tidy + 6*y_incr) + tidx + x_incr];
                
		Bs[tidy + 7*y_incr][tidx]  = B[b + N * (tidy + 7*y_incr) + tidx];
                Bs[tidy + 7*y_incr][tidx + x_incr]  = B[b + N * (tidy + 7*y_incr) + tidx + x_incr];
  */              
                
		__syncthreads();

                /* multiply the two matrices together */
                /* one thread per element of C */
#pragma unroll
                for (int k = 0; k < BLOCK_SIZE; ++k){
                        cSub += As[tidy][k] * Bs[k][tidx];
			cSub_8 += As[tidy][k] * Bs[k][tidx + x_incr];

                        cSub_1 += As[tidy + y_incr][k] * Bs[k][tidx];
			cSub_9 += As[tidy + y_incr][k] * Bs[k][tidx + x_incr];

			cSub_2 += As[tidy + 2*y_incr][k] * Bs[k][tidx];
			cSub_10 += As[tidy + 2*y_incr][k] * Bs[k][tidx + x_incr];

                        cSub_3 += As[tidy + 3*y_incr][k] * Bs[k][tidx];
			cSub_11 += As[tidy + 3*y_incr][k] * Bs[k][tidx + x_incr];                       

			cSub_4 += As[tidy + 4*y_incr][k] * Bs[k][tidx];
                        cSub_12 += As[tidy + 4*y_incr][k] * Bs[k][tidx + x_incr];  

			cSub_5 += As[tidy + 5*y_incr][k] * Bs[k][tidx];
			cSub_13 += As[tidy + 5*y_incr][k] * Bs[k][tidx + x_incr];                        

			cSub_6 += As[tidy + 6*y_incr][k] * Bs[k][tidx];
			cSub_14 += As[tidy + 6*y_incr][k] * Bs[k][tidx + x_incr];
                        
			cSub_7 += As[tidy + 7*y_incr][k] * Bs[k][tidx];
			cSub_15 += As[tidy + 7*y_incr][k] * Bs[k][tidx + x_incr];
                }
                /* synchornize before loading next sub-blocks */
                __syncthreads();
        }

        /* write back the results */
        int c = N * BLOCK_SIZE * bidy + BLOCK_SIZE * bidx;
	C[c + N * tidy + tidx] = cSub;
        C[c + N * tidy + tidx + x_incr] = cSub_8;
        
	C[c + N * (tidy + y_incr) + tidx] = cSub_1;
	C[c + N * (tidy + y_incr) + tidx + x_incr] = cSub_9;
        
	C[c + N * (tidy + 2*y_incr) + tidx] = cSub_2;
	C[c + N * (tidy + 2*y_incr) + tidx + x_incr] = cSub_10;
        
	C[c + N * (tidy + 3*y_incr) + tidx] = cSub_3;
	C[c + N * (tidy + 3*y_incr) + tidx + x_incr] = cSub_11;
        
	C[c + N * (tidy + 4*y_incr) + tidx] = cSub_4;
	C[c + N * (tidy + 4*y_incr) + tidx + x_incr] = cSub_12;
        
	C[c + N * (tidy + 5*y_incr) + tidx] = cSub_5;
	C[c + N * (tidy + 5*y_incr) + tidx + x_incr] = cSub_13;
        
	C[c + N * (tidy + 6*y_incr) + tidx] = cSub_6;
	C[c + N * (tidy + 6*y_incr) + tidx + x_incr] = cSub_14;
        
	C[c + N * (tidy + 7*y_incr) + tidx] = cSub_7;
	C[c + N * (tidy + 7*y_incr) + tidx + x_incr] = cSub_15;

}

void
mmMyOwn (dtype* A, dtype* B, dtype* C, unsigned int N)
{
	unsigned int nBlocks;


	nBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

	dim3 grid (nBlocks, nBlocks);	
	dim3 block (BLOCK_SIZE / 2, BLOCK_SIZE / 8);	

	mmMyOwnKernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmMyOwnKernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmMyOwnKernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmMyOwnKernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
	mmMyOwnKernel <<<grid, block>>> (A, B, C, N);
	hipDeviceSynchronize ();
}




void
cudaMM (dtype *A, dtype* B, dtype* C, unsigned int N, unsigned int OPT, dtype* h_C)
{
	hipEvent_t start, stop;
	float elapsedTime;

	CUDA_CHECK_ERROR (hipEventCreate (&start));
	CUDA_CHECK_ERROR (hipEventCreate (&stop));

	fprintf (stderr, "Executing test case [%d]\n", OPT);
	fprintf (stderr, "[1]: Naive | [2]: shared memory| [3]: SM 2 per thread | [4]: SM 4 per thread | [5]: SM 8 per thread | [6]: my own implementation \n");

	
	CUDA_CHECK_ERROR (hipEventRecord (start, 0));
	/* execute kernel */
	switch (OPT) {
		case 1:
			mmNaive (A, B, C, N);	
			break;
		case 2:
			mmShared (A, B, C, N);	
			break;
		case 3:
			mmShared2 (A, B, C, N);	
			break;
		case 4:
			mmShared4 (A, B, C, N);	
			break;
		case 5:
			mmShared8 (A, B, C, N);	
			break;
		case 6:
			mmMyOwn (A, B, C, N);
			break;
		default:
			mmNaive (A, B, C, N);	
	} 
	CUDA_CHECK_ERROR (hipEventRecord (stop, 0));
	CUDA_CHECK_ERROR (hipEventSynchronize (stop));
	CUDA_CHECK_ERROR (hipEventElapsedTime (&elapsedTime, start, stop));
	elapsedTime = elapsedTime / 5;

	CUDA_CHECK_ERROR (hipMemcpy (h_C, C, N * N * sizeof (dtype), 
																hipMemcpyDeviceToHost));

	fprintf (stderr, "Execution time: %f ms\n", elapsedTime);
	fprintf (stderr, "Equivalent performance: %f GFLOP/s\n", 
						1e-6 * 2 * N * N * N / elapsedTime );

	CUDA_CHECK_ERROR (hipEventDestroy (start));
	CUDA_CHECK_ERROR (hipEventDestroy (stop));

}


