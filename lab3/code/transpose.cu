#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cuda_utils.h"
#include "timer.c"

typedef float dtype;

void display(dtype *A, int N){
	for(int i = 0; i < N; i++){
		printf("%f ", A[i]);
	}
	printf("\n");
}

__global__ 
void matTrans(dtype* AT, size_t pitch_trans, dtype* A, size_t pitch, int N)  {
	/* Fill your code here */

//	FIRST METHOD
/*
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	int nThreads = gridDim.x * blockDim.x;
	int total_elements = N * N;
	int num_pass = total_elements/nThreads + 1;

	for(unsigned int i = 1; i <= num_pass; i++) {
		if(gid < total_elements)
		{
			unsigned int index = (gid / N) + ( (gid % N) * N );
		 	AT[gid] = A[index];
		}
		gid += nThreads;
	}
*/
//	SECOND METHOD
/*
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int index = x ;
	int transIndex = ( x / N) + ( (x % N) * pitch );
	if(blockIdx.x == 0)
		printf("N %3d pitch %3d BlockDim.x  %3d  BlockIdX:  %3d   x:  %3d    transIndex: %3d\n",
				blockDim.x, blockIdx.x,  x, transIndex);

	int normal_total_elems = N * N;
	int trans_total_elems = N * pitch;
	if (index < normal_total_elems && transIndex < trans_total_elems)
	{
		printf("ThreadIdx %3d\n", threadIdx.x);
		printf("Total Elements  %3d TransTotal Elements %3d BlockId %3d BlockDim %3d\n",
				 normal_total_elems, trans_total_elems, blockIdx.x, blockDim.x);
      		printf("N  %3d  Pitch  %3d  Index %3d transIndex %3d A  %3d\n", N, pitch, index, transIndex, A[index]);
		AT[transIndex] = A[index] ;
	}
*/
	//int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	//int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int tot_elems = N * N;

	//if(xIndex < N && yIndex < N)  
	if(xIndex < tot_elems)
	{  
	     int xi = xIndex % N;
	     int yi = xIndex / N;	
//	     printf("THREAD X  %3d THREAD Y  %3d X INDEX %3d TOTAL ELEMS %3d PITCH %3d\n", xi, yi, xIndex, tot_elems, pitch);
	     // update the pointer to point to the beginning of the next row  
	     dtype* rowData = (dtype*)(((char*)A) + (yi * pitch));  
	     dtype* transRowData = (dtype*)(((char*)AT)+(xi * pitch_trans));
	     transRowData[yi] = rowData[xi];  
	}  

}

void
parseArg (int argc, char** argv, int* N)
{
	if(argc == 2) {
		*N = atoi (argv[1]);
		assert (*N > 0);
	} else {
		fprintf (stderr, "usage: %s <N>\n", argv[0]);
		exit (EXIT_FAILURE);
	}
}


void
initArr (dtype* in, int N)
{
	int i;

	for(i = 0; i < N; i++) {
		in[i] = (dtype) rand () / RAND_MAX;
	}
}

void
cpuTranspose (dtype* A, dtype* AT, int N)
{
	int i, j;

	for(i = 0; i < N; i++) {
		for(j = 0; j < N; j++) {
			AT[j * N + i] = A[i * N + j];
		}
	}
}

int
cmpArr (dtype* a, dtype* b, int N)
{
	int cnt, i;

	cnt = 0;
	for(i = 0; i < N; i++) {
		if(abs(a[i] - b[i]) > 1e-6) cnt++;
	}

	return cnt;
}

void
gpuTranspose (dtype* A, dtype* AT, int N)
{
  struct stopwatch_t* timer = NULL;
  long double t_gpu, t_pcie, t_malloc;
  dtype *d_a, *d_at;
	
  /* Setup timers */
  stopwatch_init ();
  timer = stopwatch_create ();
 
  size_t pitch;
   size_t pitch_trans;

  stopwatch_start (timer);
	/* warup */
  CUDA_CHECK_ERROR( hipFree(0) );

	CUDA_CHECK_ERROR ( hipMallocPitch((void**)&d_a, &pitch, N*sizeof(dtype), N) );
	CUDA_CHECK_ERROR ( hipMallocPitch((void**)&d_at, &pitch_trans, N*sizeof(dtype), N) );
        //CUDA_CHECK_ERROR (hipMalloc ((void**) &d_at, N * N * sizeof (dtype)));
	t_malloc = stopwatch_stop (timer);
	fprintf (stderr, "hipMalloc: %Lg seconds\n", t_malloc);

	/* run your kernel here */
//	printf("PITCH : %3d \n", pitch);

//	printf("Matrix before transpose\n");
//	display(A, N*N);
//	printf("PITCH_TRANS : %3d\n", pitch_trans);

  stopwatch_start (timer);

	// copy arrays to device via PCIe
	CUDA_CHECK_ERROR ( hipMemcpy2D(d_a,pitch,A,sizeof(dtype)*N,sizeof(dtype)*N,N,hipMemcpyHostToDevice) );
	t_pcie = stopwatch_stop (timer);
	fprintf (stderr, "hipMemcpy: %Lg seconds\n", t_pcie);


	/* do not change this number */
	int nThreads = 1048576;
	int tbSize = 1024;
	int numTB = (nThreads + tbSize - 1) / tbSize;

//	dim3 threadsPerBlock(16, 16);
//	dim3 numBlocks(nThreads / threadsPerBlock.x, nThreads / threadsPerBlock.y);

	stopwatch_start (timer);

	// kernel invocation
//	matTrans <<<numBlocks, threadsPerBlock>>> (d_at, d_a, N, pitch);
	matTrans <<<numTB, tbSize>>> (d_at, pitch_trans, d_a, pitch, N);

	hipDeviceSynchronize ();
	t_gpu = stopwatch_stop (timer);
	fprintf (stderr, "GPU transpose %Lg seconds ==> %Lg billion elements per second\n", 
						t_gpu, (N / t_gpu) * 1e-9);

	// copy results back from device to host
	//CUDA_CHECK_ERROR (hipMemcpy (AT, d_at, N * N * sizeof (dtype), 
	//									hipMemcpyDeviceToHost));
	CUDA_CHECK_ERROR (hipMemcpy2D (AT,sizeof(dtype)*N,d_at,pitch_trans,sizeof(dtype)*N,N,hipMemcpyDeviceToHost));
	// copy results back from device to host
        //CUDA_CHECK_ERROR (hipMemcpy (AT, d_at, N * N * sizeof (dtype), 
        //                                                                        hipMemcpyDeviceToHost));
//	printf("Matrix after transpose\n");
//	display(AT, N*N);

	// free memory on device
	CUDA_CHECK_ERROR (hipFree (d_a));
	CUDA_CHECK_ERROR (hipFree (d_at));

}

int 
main(int argc, char** argv)
{
  /* variables */
	dtype *A, *ATgpu, *ATcpu;
  int err;

	int N;

  struct stopwatch_t* timer = NULL;
  long double t_cpu;


	N = -1;
	parseArg (argc, argv, &N);

  /* input and output matrices on host */
  /* output */
  ATcpu = (dtype*) malloc (N * N * sizeof (dtype));
  ATgpu = (dtype*) malloc (N * N * sizeof (dtype));

  /* input */
  A = (dtype*) malloc (N * N * sizeof (dtype));

	initArr (A, N * N);

	/* GPU transpose kernel */
	gpuTranspose (A, ATgpu, N);

  /* Setup timers */
  stopwatch_init ();
  timer = stopwatch_create ();

	stopwatch_start (timer);
  /* compute reference array */
	cpuTranspose (A, ATcpu, N);
  t_cpu = stopwatch_stop (timer);
  fprintf (stderr, "Time to execute CPU transpose kernel: %Lg secs\n",
           t_cpu);

  /* check correctness */
	err = cmpArr (ATgpu, ATcpu, N * N);
	if(err) {
		fprintf (stderr, "Transpose failed: %d\n", err);
	} else {
		fprintf (stderr, "Transpose successful\n");
	}

	free (A);
	free (ATgpu);
	free (ATcpu);

  return 0;
}
