#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "cuda_utils.h"
#include "timer.c"


typedef float dtype;


void
parseArg (int argc, char** argv, int* N)
{
	if(argc == 2) {
		*N = atoi (argv[1]);
		assert (*N > 0);
	} else {
		fprintf (stderr, "usage: %s <N>\n", argv[0]);	
		exit (EXIT_FAILURE);
	}
}


void
cpuSaxpy (dtype a, dtype* x, dtype* y, int N)
{
	int i;
	
	for(i = 0;i < N; i++) {
		y[i] = a * x[i] + y[i];
	}
}


int
cmpArr (dtype* a, dtype* b, int N)
{
	int cnt, i;

	cnt = 0;
	for(i = 0; i < N; i++) {
		if(abs(a[i] - b[i]) > 1e-6) cnt++;
	}

	return cnt;
}


__global__ void
saxpy (dtype a, dtype* x, dtype* y, int N)
{
	/* fill in your code here */
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	int nThreads = gridDim.x * blockDim.x;
	int num_pass = N/nThreads + 1;

	for(int i=0; i< num_pass; i++){
		if(gid < N) y[gid] = a * x[gid] + y[gid];	
		gid +=nThreads;
	}
}


void
gpuSaxpy (dtype a, dtype* h_x, dtype* h_y, int N)
{
	dtype *d_x, *d_y;
	int nThreads, tbSize, numTB;

	struct stopwatch_t* timer;
	long double t_gpu, t_pcie, t_malloc;


	// create timers
	stopwatch_init ();
	timer = stopwatch_create ();
	assert (timer);

	stopwatch_start (timer);
	// allocate memory on device
	CUDA_CHECK_ERROR (hipMalloc ((void**) &d_x, N * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMalloc ((void**) &d_y, N * sizeof (dtype)));
	t_malloc = stopwatch_stop (timer);
	fprintf (stderr, "hipMalloc: %Lg seconds\n", t_malloc);


	stopwatch_start (timer);
	// copy arrays to device via PCIe
	CUDA_CHECK_ERROR (hipMemcpy (d_x, h_x, N * sizeof (dtype), 
										hipMemcpyHostToDevice));
	CUDA_CHECK_ERROR (hipMemcpy (d_y, h_y, N * sizeof (dtype), 
										hipMemcpyHostToDevice));
	t_pcie = stopwatch_stop (timer);
	fprintf (stderr, "hipMemcpy: %Lg seconds\n", t_pcie);


	/* do not change this number */
	nThreads = 1048576;
	tbSize = 256;
	numTB = (nThreads + tbSize - 1) / 256;

	stopwatch_start (timer);
	// kernel invocation
	saxpy <<<numTB, tbSize>>> (a, d_x, d_y, N);
	hipDeviceSynchronize ();
	t_gpu = stopwatch_stop (timer);
	fprintf (stderr, "SAXPY: %Lg seconds ==> %Lg billiion elements per second\n", 
						t_gpu, (N / t_gpu) * 1e-9);

	// copy results back from device to host
	CUDA_CHECK_ERROR (hipMemcpy (h_y, d_y, N * sizeof (dtype), 
										hipMemcpyDeviceToHost));

	// free memory on device
	CUDA_CHECK_ERROR (hipFree (d_x));
	CUDA_CHECK_ERROR (hipFree (d_y));
}


void
initArr (dtype* in, int N)
{
	int i;
	
	for(i = 0; i < N; i++) {
		in[i] = (dtype) rand () / RAND_MAX;
	}
}

void initA (dtype* a)
{
	*a = (dtype) rand () / RAND_MAX;
}

void copyArr (dtype* dst, dtype* src, int N)
{
	int i;

	for(i = 0;i < N; i++) {
		dst[i] = src[i];
	}
}


int 
main (int argc, char** argv)
{	
	dtype *x, *y, *y_cpu;
	dtype a;

	int N;
	int err;

	struct stopwatch_t* timer = NULL;
	long double t_cpu;


	N = -1;
	parseArg (argc, argv, &N);

	/* create host data structures */
	x = (dtype*) malloc (N * sizeof (dtype));
	y = (dtype*) malloc (N * sizeof (dtype));
	y_cpu = (dtype*) malloc (N * sizeof (dtype));

	/* initialize arrays */
	initArr (x, N);
	initArr (y, N);
	copyArr (y_cpu, y, N);
	initA (&a);

	/* create timers */
	stopwatch_init ();
	timer = stopwatch_create ();
	assert (timer);

	/* call function for GPU SAXPY */
	/* y = ax + y */
	gpuSaxpy (a, x, y, N);

	stopwatch_start (timer);
	/* verify results on CPU */
	cpuSaxpy (a, x, y_cpu, N);
	t_cpu = stopwatch_stop (timer);
	fprintf (stderr, "CPU: %Lg seconds\n", t_cpu);

	/* compare answers */
	err = cmpArr (y, y_cpu, N);
	if(!err) {
		fprintf (stderr, "Correct answer\n");
	} else {
		fprintf (stderr, "Wrong answers: %d out of %d\n", err, N);
	}	

	free (x);
	free (y);
	free (y_cpu);
	
	return 0;
}
